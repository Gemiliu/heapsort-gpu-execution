#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "timer.h"
#include "check.h"
#include <iostream>
#include <math.h>

static const int blockSize = 127; // array size
static const int iterations = 10; // number of iterations
int countBlocks = 250;

inline void cudaCheck(const hipError_t &err, const std::string &mes) {
	if (err != hipSuccess) {
		std::cout << (mes + " - " + hipGetErrorString(err)) << std::endl;
		exit(EXIT_FAILURE);
	}
}

__device__ void swap(float *a, float *b) {
	const float t = *a;
	*a = *b;
	*b = t;
}

__device__ void maxHeapify(float *maxHeap, int heapSize, int idx) {
	int largest = idx;  // Initialize largest as root
	int left = (idx << 1) + 1;  // left = 2*idx + 1
	int right = (idx + 1) << 1; // right = 2*idx + 2

	// See if left child of root exists and is greater than root
	if (left < heapSize && maxHeap[left] > maxHeap[largest]) {
		largest = left;
	}

	// See if right child of root exists and is greater than
	// the largest so far
	if (right < heapSize && maxHeap[right] > maxHeap[largest]) {
		largest = right;
	}

	// Change root, if needed
	if (largest != idx) {
		swap(&maxHeap[largest], &maxHeap[idx]);
		maxHeapify(maxHeap, heapSize, largest);
	}
}

// A utility function to create a max heap of given capacity
__device__ void createAndBuildHeap(float *array, int size) {
	// Start from bottommost and rightmost internal mode and heapify all
	// internal modes in bottom up way
	for (int i = (size - 2) / 2; i >= 0; --i) {
		maxHeapify(array, size, i);
	}
}

__global__ void heapSortKernel(float *iA, int size) {
	//A = A + blockIdx.x * blockSize;
	iA = iA + blockIdx.x * blockSize;
	__shared__ float A[blockSize];
	for (int i = threadIdx.x; i < blockSize; i += blockDim.x) {
		A[i] = iA[i];
	}
	__syncthreads();
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadIdx.x == 0) {
		// Build a heap from the input data.
		createAndBuildHeap(A, size);

		// Repeat following steps while heap size is greater than 1.
		// The last element in max heap will be the minimum element
		int changedSizeOfHeap = size;
		while (changedSizeOfHeap > 1) {
			// The largest item in Heap is stored at the root. Replace
			// it with the last item of the heap followed by reducing the
			// size of heap by 1.
			swap(A, &A[changedSizeOfHeap - 1]);
			--changedSizeOfHeap;  // Reduce heap size

			// Finally, heapify the root of tree.
			maxHeapify(A, changedSizeOfHeap, 0);
		}
	}
	for (int i = threadIdx.x; i < blockSize; i += blockDim.x) {
		iA[i] = A[i];
	}
}

int main(int argc,char *argv[]) {
	if (argc == 2) {
		countBlocks = atoi(argv[1]);
	}
	std::cout << "count blocks = " << countBlocks << std::endl;

	hipError_t err = hipSuccess;
	// Print the vector length to be used, and compute its size
	int numElements = blockSize * countBlocks;
	size_t size = numElements * sizeof(float);

	// Allocate the host input vector A
	float *h_A = (float *)malloc(size);
	if (h_A == NULL) {
		std::cout << "Failed to allocate host vectors!" << std::endl;
		exit(EXIT_FAILURE);
	}
	// Allocate the device input vector A
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);
	cudaCheck(err, "failed to allocated device vector A");

	double proccesingTime = 0.0f;
	double commonProccesingTime = 0.0f;
	double proccesingTimeWithCopying = 0.0f;
	for (int k = 0; k < iterations; ++k) {
		Time timer;
		timer.begin("common");
		// Initialize the host input vectors
		for (int i = 0; i < numElements; ++i) {
			//h_A[i] = rand()/(float)RAND_MAX;
			h_A[i] = rand() % 1000 + 1;
		}
		/*for (int i = 0; i < countBlocks; ++i) {
			for (int j = 0; j < blockSize; ++j) {
				const int index = i * blockSize + j;
				std::cout << h_A[index] << " ";
			}
			std::cout << std::endl;
		}*/

		timer.begin("with copying");
		err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
		cudaCheck(err, "failed to copy vector A to device");

		// Launch the CUDA Kernel
		int threadsPerBlock = 32;
		int blocksPerGrid = countBlocks;
		timer.begin("sort");
		heapSortKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, blockSize);
		hipDeviceSynchronize();
		timer.end("sort");
		err = hipGetLastError();
		cudaCheck(err, "failed to launch kernel");

		err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
		cudaCheck(err, "failed to copy vector A to host");
		timer.end("with copying");
		/*for (int i = 0; i < countBlocks; ++i) {
			for (int j = 0; j < blockSize; ++j) {
				const int index = i * blockSize + j;
				std::cout << h_A[index] << " ";
			}
			std::cout << std::endl;
		}*/
		// Verify that the result vector is correct
		for (int i = 0; i < countBlocks; ++i) {
			for (int j = 0; j < (blockSize - 1); ++j) {
				const int index = i * blockSize + j;
				if ((h_A[index] - h_A[index + 1]) > 0.00001) {
					std::cout << "Result verification failed at element " << i << "!" << std::endl;
					exit(EXIT_FAILURE);
				}
			}
		}
		timer.end("common");
		proccesingTime +=timer.getTimeSecFloat("sort");
		proccesingTimeWithCopying +=timer.getTimeSecFloat("with copying");
		commonProccesingTime += timer.getTimeSecFloat("common");
	}
	std::cout << "Test PASSED" << std::endl;
	const float avgProccesingTime = proccesingTime / iterations;
	const float avgProccesingTimeWithCopying = proccesingTimeWithCopying / iterations;
	std::cout << "avg proccesing time = " << avgProccesingTime << " sec" << std::endl;
	std::cout << "avg proccesing time(with copying) = " << avgProccesingTimeWithCopying << " sec" << std::endl;
	const int countOperations = countBlocks * (blockSize * log2f(blockSize));
	std::cout << "Computational throughput = " << countOperations / (avgProccesingTime * 10e6) << " MB/s" << std::endl;
	std::cout << "Computational throughput(with copying) = " << countOperations / (avgProccesingTimeWithCopying * 10e6) << " MB/s" << std::endl;

    // Free device global memory
    err = hipFree(d_A);
	cudaCheck(err, "failed to free device vector A");
	// Free host memory
    free(h_A);
    err = hipDeviceReset();
  	cudaCheck(err, "failed to deinitialize the device");

    std::cout << "Done." << std::endl;
    return 0;
}
